#include "hip/hip_runtime.h"
#include<cstdint>
#include<cassert>

__global__  
void radixSort(int16_t * v, uint32_t * ind, uint32_t size) {
     
  constexpr int d = 8, w = 16;
  constexpr int sb = 1<<d;

  constexpr int MaxSize = 256*32;
  __shared__ uint32_t ind2[MaxSize];
  __shared__ uint32_t c[sb];
  __shared__ uint32_t firstNeg;    

  assert(size<=MaxSize);  // for multiple blocks this is not correct
  assert(blockDim.x==sb);  

  assert(blockIdx.x==0);
  // int first = blockDim.x * blockIdx.x + threadIdx.x;

  firstNeg=0;

  auto a = v; // later add offset
  auto j = ind; // later add offset
  auto k = ind2;

  int32_t first = threadIdx.x;
  for (auto i=first; i<size; i+=blockDim.x)  j[i]=i;
  __syncthreads();


  for (int p = 0; p < w/d; ++p) {
    c[threadIdx.x]=0;
    __syncthreads();

    // fill bins
    for (auto i=first; i<size; i+=blockDim.x) 
      atomicAdd(&c[(a[j[i]] >> d*p)&(sb-1)],1);
    __syncthreads();

    // prefix scan to be optimized...
    if (threadIdx.x==0)
      for (int i = 1; i < sb; ++i) c[i] += c[i-1];
    __syncthreads();

    // broadcast
    if (threadIdx.x==0)
    for (int i=size-first-1; i>=0; i--) { // =blockDim.x) {
      auto ik = atomicSub(&c[(a[j[i]] >> d*p)&(sb-1)],1);
      k[ik-1] = j[i];
    }
    __syncthreads();

    // swap (local, ok)
    auto t=j;j=k;k=t;
  }

  // w/d is even so ind is correct
  assert(j==ind);
  __syncthreads();

  

  // now move negative first...
  // find first negative
  for (auto i=first; i<size-1; i+=blockDim.x) {
    // if ( (int(a[ind[i]])*int(a[ind[i+1]])) <0 ) firstNeg=i+1;
   if ( (a[ind[i]]^a[ind[i+1]]) < 0 ) firstNeg=i+1; 
  }
  
  __syncthreads();
  assert(firstNeg>0);

  auto ii=first;
  for (auto i=firstNeg+threadIdx.x; i<size; i+=blockDim.x)  { ind2[ii] = ind[i]; ii+=blockDim.x; }
  __syncthreads();
  ii= size-firstNeg +threadIdx.x;
  assert(ii>=0);
  for (auto i=first;i<firstNeg;i+=blockDim.x)  { ind2[ii] = ind[i]; ii+=blockDim.x; }
  __syncthreads();
  for (auto i=first; i<size; i+=blockDim.x) ind[i]=ind2[i];

  
}


#include "cuda/api_wrappers.h"

#include <iomanip>
#include <memory>
#include <algorithm>
#include <chrono>
#include<random>


#include<cassert>
#include<iostream>

int main() {

  auto start = std::chrono::high_resolution_clock::now();
  auto delta = start - start;

	if (cuda::device::count() == 0) {
		std::cerr << "No CUDA devices on this system" << "\n";
		exit(EXIT_FAILURE);
	}

        auto current_device = cuda::device::current::get(); 


  constexpr int N=256*32;
  int16_t v[N];
  uint32_t ind[N];

  std::cout << "Will sort " << N << " shorts" << std::endl;

  for (int i = 0; i < N; i++) {
    v[i]=i%32768; if(i%2) v[i]=-v[i];
  }

  auto v_d = cuda::memory::device::make_unique<int16_t[]>(current_device, N);
  auto ind_d = cuda::memory::device::make_unique<uint32_t[]>(current_device, N);
  cuda::memory::copy(v_d.get(), v, 2*N);

   int threadsPerBlock =256;
   int blocksPerGrid = 1;
   delta -= (std::chrono::high_resolution_clock::now()-start);
   cuda::launch(
                radixSort,
                { blocksPerGrid, threadsPerBlock },
                v_d.get(),ind_d.get(),N
        );


   delta += (std::chrono::high_resolution_clock::now()-start);
   std::cout <<"cuda computation took "
              << std::chrono::duration_cast<std::chrono::milliseconds>(delta).count()
              << " ms" << std::endl;

//  cuda::memory::copy(v, v_d.get(), 2*N);
 cuda::memory::copy(ind, ind_d.get(), 4*N);

 std::cout << v[ind[0]] << ' ' << v[ind[1]] << ' ' << v[ind[2]] << std::endl;
   std::cout << v[ind[3]] << ' ' << v[ind[10]] << ' ' << v[ind[N-1000]] << std::endl;
  std::cout << v[ind[N/2-1]] << ' ' << v[ind[N/2]] << ' ' << v[ind[N/2+1]] << std::endl;
 for (int i = 1; i < N; i++) {
    if (v[ind[i]]<v[ind[i-1]])
      std::cout << "not ordered at " << ind[i] << " : "
		<< v[ind[i]] <<' '<< v[ind[i-1]] << std::endl;
 }
  return 0;
}
