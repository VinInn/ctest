
#include <hip/hip_runtime.h>
__global__
void glo(int * x, int * y) {
   atomicAdd(x+3,1);
}

__global__
void blo(int * x, int *  y) {
   atomicAdd_block(x+3,1);
}

__global__
void sha(int * x, int *  y) {
   __shared__ int c[1024];
   atomicAdd(c+3,1);
   x[3] = c[3];
}


__global__
void shablo(int * x, int *  y) {
   __shared__ int c[1024];
   atomicAdd_block(c+3,1);
   x[3] = c[3];
}

