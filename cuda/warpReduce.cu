
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include <algorithm>

__global__ void warpReduce() {
    int laneId = threadIdx.x & 0x1f;
    // Seed starting value as inverse lane ID
    int value = 31 - laneId;

    // Use XOR mode to perform butterfly reduction
    for (int i=16; i>=1; i/=2)
        value += __shfl_xor_sync(0xffffffff, value, i, 32);

    // "value" now contains the sum across all threads
    printf("Thread %d final value = %d\n", threadIdx.x, value);
}

__global__ void warpMin() {
    int laneId = threadIdx.x & 0x1f;
    // Seed starting value as inverse lane ID
    int value = 15 - laneId;

    // Use XOR mode to perform butterfly reduction
    for (int i=16; i>=1; i/=2)
        value = std::min(value,__shfl_xor_sync(0xffffffff, value, i, 32));

    // "value" now contains the sum across all threads
    printf("Thread %d final value = %d\n", threadIdx.x, value);
}


int main() {
    warpReduce<<< 1, 32 >>>();
    warpMin<<< 1, 32 >>>();
    hipDeviceSynchronize();

    return 0;
}
