
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cassert>

template<typename T>
__device__
void 
__forceinline__
warpPrefixScan(T * c, uint32_t i) {
   auto x = c[i];
   auto laneId = threadIdx.x & 0x1f;
   #pragma unroll
   for( int offset = 1 ; offset < 32 ; offset <<= 1 ) {
     auto y = __shfl_up_sync(0xffffffff,x, offset);
     if(laneId >= offset) x += y;
   }
   c[i] = x;
}

// limited to 32*32 elements....
template<typename T>
__device__
void
__forceinline__
blockPrefixScan(T * c, uint32_t size, T* ws) {
  assert(size<=1024);
  assert(0==blockDim.x%32);

  auto first = threadIdx.x;

  for (auto i=first; i<size; i+=blockDim.x) {
    warpPrefixScan(c,i);
    auto laneId = threadIdx.x & 0x1f;
    auto warpId = i/32;
    assert(warpId<32);
    if (31==laneId) ws[warpId]=c[i];
  }
  __syncthreads();
  if (size<=32) return;
  if (threadIdx.x<32) warpPrefixScan(ws,threadIdx.x);
  __syncthreads();
  for (auto i=first+32; i<size; i+=blockDim.x) {
    auto warpId = i/32;
    c[i]+=ws[warpId-1];
  }
  __syncthreads();
}

template<typename T>
__global__
void testPrefixScan(uint32_t size) {

  __shared__ T ws[32];
  __shared__ T c[1024];
  auto first = threadIdx.x;
  for (auto i=first; i<size; i+=blockDim.x) c[i]=1;
  __syncthreads();

  blockPrefixScan(c, size, ws);

  assert(1==c[0]);
  for (auto i=first+1; i<size; i+=blockDim.x) {
    if (c[i]!=c[i-1]+1) printf("failed %d %d %d: %d %d\n",size, i, blockDim.x, c[i],c[i-1]);
    assert(c[i]==c[i-1]+1); assert(c[i]==i+1);
  }
}


template<typename T>
__global__
void testWarpPrefixScan(uint32_t size) {
  assert(size<=32);
  __shared__ T c[1024];
  auto i = threadIdx.x;
  c[i]=1;
  __syncthreads();

  warpPrefixScan(c,i);
 __syncthreads();

  assert(1==c[0]);
  if(i!=0) {
    if (c[i]!=c[i-1]+1) printf("failed %d %d %d: %d %d\n",size, i, blockDim.x, c[i],c[i-1]);
    assert(c[i]==c[i-1]+1); assert(c[i]==i+1);
  }
}


#include<iostream>
int main() {

  std::cout << "warp 32" << std::endl;
  testWarpPrefixScan<int><<<1,32>>>(32);
  hipDeviceSynchronize();
  std::cout << "warp 16" << std::endl;
  testWarpPrefixScan<int><<<1,32>>>(16);
  hipDeviceSynchronize();
  std::cout << "warp 5" << std::endl;
  testWarpPrefixScan<int><<<1,32>>>(5);
  hipDeviceSynchronize();

  for(int bs=32; bs<=1024; bs+=32) {
  std::cout << "bs " << bs << std::endl;
  for (int j=1;j<=1024; ++j) {
   std::cout << j << std::endl;
   testPrefixScan<uint16_t><<<1,bs>>>(j);
  hipDeviceSynchronize();
//   testPrefixScan<float><<<1,bs>>>(j);
//  cudaDeviceSynchronize();
  }}
  hipDeviceSynchronize();

  return 0;
}
