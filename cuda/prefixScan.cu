#include "hip/hip_runtime.h"
#include "prefixScan.h"


template<typename T>
__global__
void testPrefixScan(uint32_t size) {

  __shared__ T ws[32];
  __shared__ T c[1024];
  auto first = threadIdx.x;
  for (auto i=first; i<size; i+=blockDim.x) c[i]=1;
  __syncthreads();

  blockPrefixScan(c, size, ws);

  assert(1==c[0]);
  for (auto i=first+1; i<size; i+=blockDim.x) {
    if (c[i]!=c[i-1]+1) printf("failed %d %d %d: %d %d\n",size, i, blockDim.x, c[i],c[i-1]);
    assert(c[i]==c[i-1]+1); assert(c[i]==i+1);
  }
}


template<typename T>
__global__
void testWarpPrefixScan(uint32_t size) {
  assert(size<=32);
  __shared__ T c[1024];
  auto i = threadIdx.x;
  c[i]=1;
  __syncthreads();

  warpPrefixScan(c,i,0xffffffff);
 __syncthreads();

  assert(1==c[0]);
  if(i!=0) {
    if (c[i]!=c[i-1]+1) printf("failed %d %d %d: %d %d\n",size, i, blockDim.x, c[i],c[i-1]);
    assert(c[i]==c[i-1]+1); assert(c[i]==i+1);
  }
}


#include<iostream>
int main() {

  std::cout << "warp 32" << std::endl;
  testWarpPrefixScan<int><<<1,32>>>(32);
  hipDeviceSynchronize();
  std::cout << "warp 16" << std::endl;
  testWarpPrefixScan<int><<<1,32>>>(16);
  hipDeviceSynchronize();
  std::cout << "warp 5" << std::endl;
  testWarpPrefixScan<int><<<1,32>>>(5);
  hipDeviceSynchronize();

  for(int bs=32; bs<=1024; bs+=32) {
//  std::cout << "bs " << bs << std::endl;
  for (int j=1;j<=1024; ++j) {
//   std::cout << j << std::endl;
   testPrefixScan<uint16_t><<<1,bs>>>(j);
   hipDeviceSynchronize();
   testPrefixScan<float><<<1,bs>>>(j);
  hipDeviceSynchronize();
  }}
  hipDeviceSynchronize();

  return 0;
}
