#include "hip/hip_runtime.h"
#include "CUDATask.h"


__global__ void one(int32_t *d_in, int32_t *d_out,  int32_t n) {

  auto init = [&](int32_t iWork) {
    // standard loop  (iWork instead of blockIdx.x)
    auto first = iWork * blockDim.x + threadIdx.x;
   for (int i=first; i<n; i+=gridDim.x*blockDim.x) { d_in[i]=-1; d_out[i]=-5;}
   d_in[3333]=-4;  // touch it everywhere
   if (15==d_in[1234]) d_in[1234]=33;
   if (15==d_out[200234]) d_out[200234]=33;
  };

  init(blockIdx.x);
}


__global__ void two(int32_t *d_in, int32_t *d_out,  int32_t n) {

  auto setIt = [&](int32_t iWork) {
    // standard loop  (iWork instead of blockIdx.x)
    auto first = iWork * blockDim.x + threadIdx.x;
   for (int i=first; i<n; i+=gridDim.x*blockDim.x) d_in[i]=5;
   d_in[5324]=4;  // should fail
   if (15==d_in[10234]) d_in[10234]=33;
   if (15==d_out[10234]) d_out[10234]=33;
  };

  setIt(blockIdx.x);

}

__global__ void three(int32_t *d_in, int32_t *d_out,  int32_t n) {

  auto testIt1 = [&](int32_t iWork) {
    // standard loop  (iWork instead of blockIdx.x)
    auto first = (gridDim.x-iWork-1)  * blockDim.x + threadIdx.x;
    for (int i=first; i<n; i+=gridDim.x*blockDim.x) if (5==d_in[i]) d_out[i]=5;
  };

  testIt1(blockIdx.x);

}

template<int N>
__global__ void testTask(int32_t *d_in, int32_t *d_out,  int32_t n, CUDATask * task1, CUDATask * task2) {

  auto voidTail = [](){};
  auto zero1 = [&](){task1->zero();};
  auto zero2 = [&](){task2->zero();};


  auto init = [&](int32_t iWork) {
    // standard loop  (iWork instead of blockIdx.x)
    auto first = iWork * blockDim.x + threadIdx.x;
   for (int i=first; i<n; i+=gridDim.x*blockDim.x) { d_in[i]=-1; d_out[i]=-5;}  
   d_in[3333]=-4;  // touch it everywhere
   if (15==d_in[1234]) d_in[1234]=33;
   if (15==d_out[200234]) d_out[200234]=33;

  };


  auto setIt = [&](int32_t iWork) {
    // standard loop  (iWork instead of blockIdx.x)
    auto first = iWork * blockDim.x + threadIdx.x;
   for (int i=first; i<n; i+=gridDim.x*blockDim.x) d_in[i]=5;
   d_in[5324]=4;  // should fail
   if (15==d_in[10234]) d_in[10234]=33;
   if (15==d_out[10234]) d_out[10234]=33;

  };

  auto testIt1 = [&](int32_t iWork) {
    // standard loop  (iWork instead of blockIdx.x)
    auto first = (gridDim.x-iWork-1)  * blockDim.x + threadIdx.x;
    for (int i=first; i<n; i+=gridDim.x*blockDim.x) if (5==d_in[i]) d_out[i]=5;
  };

  task1->doit(init,zero2);
  task2->doit(setIt,zero1);
  task1->doit(testIt1,zero2);

}


__global__ void verify(int32_t *d_out,  int32_t n) {
   auto first = blockIdx.x * blockDim.x + threadIdx.x;
   for (int i=first; i<n; i+=gridDim.x*blockDim.x) if (5!=d_out[i]) printf("failed %d %d/%d\n",i,blockIdx.x,threadIdx.x);

}


#include <iostream>

#include "cudaCheck.h"
#include "requireDevices.h"
#include <chrono>

using namespace std::chrono;


int main() {

  cms::cudatest::requireDevices();

  int32_t *d_in;
  int32_t *d_out1;
  int32_t *d_out2;

  int32_t num_items = 1000*1000;

  cudaCheck(hipMalloc(&d_in, num_items * sizeof(uint32_t)));
  cudaCheck(hipMalloc(&d_out1, num_items * sizeof(uint32_t)));
  cudaCheck(hipMalloc(&d_out2, num_items * sizeof(uint32_t)));

  auto nthreads = 256;
  auto nblocks = (num_items + nthreads - 1) / nthreads;

  CUDATask * task1;
  cudaCheck(hipMalloc(&task1, sizeof(CUDATask)));
  cudaCheck(hipMemset(task1, 0, sizeof(CUDATask)));

  CUDATask * task2;
  cudaCheck(hipMalloc(&task2, sizeof(CUDATask)));
  cudaCheck(hipMemset(task2, 0, sizeof(CUDATask)));


  cudaCheck(hipMemset(d_in, 0, num_items*sizeof(int32_t)));
  cudaCheck(hipMemset(d_out1, 0, num_items*sizeof(int32_t)));

  {
  std::cout << "scheduling " << nblocks << " blocks of " << nthreads << " threads"<< std::endl;
  hipDeviceSynchronize();
  high_resolution_clock::time_point t1 = high_resolution_clock::now();
  one<<<nblocks, nthreads, 0>>>(d_in, d_out1, num_items);
  two<<<nblocks, nthreads, 0>>>(d_in, d_out1, num_items);
  three<<<nblocks, nthreads, 0>>>(d_in, d_out1, num_items);
  hipDeviceSynchronize();
  high_resolution_clock::time_point t2 = high_resolution_clock::now();

  cudaCheck(hipGetLastError());
  verify<<<nblocks, nthreads, 0>>>(d_out1, num_items);
  cudaCheck(hipGetLastError());
  hipDeviceSynchronize();
  auto delta = duration_cast<duration<double>>(t2 - t1).count();
  std::cout << "three kernels took " << delta << std::endl;
  }

  cudaCheck(hipMemset(d_in, 0, num_items*sizeof(int32_t)));
  cudaCheck(hipMemset(d_out1, 0, num_items*sizeof(int32_t)));

  {
  std::cout << "scheduling " << nblocks << " blocks of " << nthreads << " threads"<< std::endl;
  hipDeviceSynchronize();
  high_resolution_clock::time_point t1 = high_resolution_clock::now();
  testTask<1> <<<nblocks, nthreads, 0>>>(d_in, d_out1, num_items, task1, task2);
  hipDeviceSynchronize();
  high_resolution_clock::time_point t2 = high_resolution_clock::now();
  cudaCheck(hipGetLastError());
  verify<<<nblocks, nthreads, 0>>>(d_out1, num_items);
  cudaCheck(hipGetLastError());
  hipDeviceSynchronize(); 
  auto delta = duration_cast<duration<double>>(t2 - t1).count();
  std::cout << "task kernel took " << delta << std::endl;
  }

  {
  nblocks /= 32;
  std::cout << "scheduling " << nblocks << " blocks of " << nthreads << " threads"<< std::endl;
  hipDeviceSynchronize();
  high_resolution_clock::time_point t1 = high_resolution_clock::now();
  testTask<2> <<<nblocks, nthreads, 0>>>(d_in, d_out1, num_items, task1, task2);
  hipDeviceSynchronize();
  high_resolution_clock::time_point t2 = high_resolution_clock::now();
  cudaCheck(hipGetLastError());
  verify<<<nblocks, nthreads, 0>>>(d_out1, num_items);
  cudaCheck(hipGetLastError());
  hipDeviceSynchronize();
  auto delta = duration_cast<duration<double>>(t2 - t1).count();
  std::cout << "task kernel took " << delta << std::endl;
  }

  return 0;
};
 
