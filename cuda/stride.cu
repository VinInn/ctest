#include "hip/hip_runtime.h"
#include<cassert>
#include<cstdint>
#include<cmath>
#include<random>
#include<vector>

template<int STRIDE>
__global__
void nn(float const * __restrict__ z, float const * __restrict__ w, uint32_t * __restrict__ nns, int ntot, float eps) {
    // this part is actually run STRIDE times for each "z"
    auto ldx = blockIdx.x * blockDim.x + threadIdx.x;
    auto idx = ldx/STRIDE;
    auto first = ldx - idx*STRIDE;
    assert(first<STRIDE);
    // usual loop uder the assumption ntot is not kown on HOST side
    auto incr = (blockDim.x * gridDim.x)/STRIDE;
    for (auto j = idx; j < ntot; j += incr) {

      // combinatorial loop  (n^2)
      // in reality it should be limited using a Histogram, KDTree or similar
      // here we parallelize. for each "z[j]" STRIDE threads are actually used
      auto k = j+ 1+first;
      for (;k < ntot; k +=STRIDE) {
        if (
             fabs(z[j]-z[k]) < eps && 
             fabs(w[j]-w[k]) < eps
           ) {
          atomicAdd(&nns[j],1);
          atomicAdd(&nns[k],1);
        }
      }  // inner loop k
    } // outer loop j

}

#include <cuda/api_wrappers.h>
#include<iostream>


constexpr uint32_t NTOT = 1024*8;

template<int STRIDE>
void go(float * z_d, float * w_d, uint32_t * nss_d) {

  auto nt = 128;
  auto nb = 1024*STRIDE;

  nn<STRIDE><<<nb,nt>>>(z_d,w_d,nss_d,NTOT,0.1f);

}


int main() {

  if (cuda::device::count() == 0) {
    std::cerr << "No CUDA devices on this system" << "\n";
    exit(EXIT_FAILURE);
  }

  auto current_device = cuda::device::current::get();

  auto z_d = cuda::memory::device::make_unique<float[]>(current_device, NTOT);
  auto w_d = cuda::memory::device::make_unique<float[]>(current_device, NTOT);
  auto nns_d = cuda::memory::device::make_unique<uint32_t[]>(current_device, NTOT);

  for (int i=0; i<16; ++i) {

  hipMemset(nns_d.get(),0,NTOT*sizeof(uint32_t));

  std::vector<float> z_h(NTOT);  // for "unclear" reasons this is now zeroed...
  std::mt19937 reng;
  std::uniform_real_distribution<float> rgen(-1.,1.);

  for (auto & z : z_h) z = rgen(reng);
  cuda::memory::copy(z_d.get(),z_h.data(),sizeof(float)*z_h.size());
  for (auto & z : z_h) z = rgen(reng);
  cuda::memory::copy(w_d.get(),z_h.data(),sizeof(float)*z_h.size());


  go<1>(z_d.get(),w_d.get(),nns_d.get());
  go<2>(z_d.get(),w_d.get(),nns_d.get());
  go<4>(z_d.get(),w_d.get(),nns_d.get());
  go<8>(z_d.get(),w_d.get(),nns_d.get());
  go<16>(z_d.get(),w_d.get(),nns_d.get());


  }
  hipDeviceSynchronize();

  return 0;

}
