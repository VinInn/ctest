#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include<cstdio>

__global__
void bar(int * i) {
  extern __shared__  unsigned char shared_mem[];
  shared_mem[threadIdx.x]=i[threadIdx.x];
  __syncthreads();
  printf("bar %d\n", shared_mem[0]);
}

struct Large {
  int v[100];
};

__global__
void huge(int * i,
  int * a1,
  int * a2,
  int * a3,
  int * a4,
  int * a5,
  int * a6,
  int * a7,
  int * a8,
  Large l1, Large l2, Large l3
) {
  extern __shared__  unsigned char shared_mem[];
  shared_mem[threadIdx.x]=i[threadIdx.x];
  __syncthreads();
  printf("bar %d %d\n", shared_mem[0], l1.v[3]);
}


/*
__global__
void crash() {
  bar<<<1,1>>>();
  hipDeviceSynchronize();
}
*/

#include "cudaCheck.h"
void wrapper() {
  int a[10]; a[0]=4;
  int * d;
  hipMalloc(&d,40);
  hipMemcpyAsync(d,a,40,hipMemcpyHostToDevice,0);
  bar<<<1,1,1024,0>>>(d);
  cudaCheck(hipGetLastError());
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
  Large l1, l2, l3;
  l1.v[3]=5;
  huge<<<1,1,1024>>>(d, d,d,d,d, d,d,d,d,l1,l2,l3);
  cudaCheck(hipGetLastError());
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
}


#include<iostream>
struct Me {

  Me() {
   std::cout << "Loaded" << std::endl;
   wrapper();
  }

};


Me me;
