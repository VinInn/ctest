#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include<cstdio>

__global__
void bar() {
  printf("bar\n");
}

__global__
void crash() {
  bar<<<1,1>>>();
  hipDeviceSynchronize();
}


#include "cudaCheck.h"
void wrapper() {
//  bar<<<1,1>>>();
  cudaCheck(hipGetLastError());
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
}


#include<iostream>
struct Me {

  Me() {
   std::cout << "Loaded" << std::endl;
   wrapper();
  }

};


Me me;
