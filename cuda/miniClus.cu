#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <algorithm>
#include <cassert>

__global__
void perm(uint16_t const __restrict__ * x, int * id,  int nt) {

    if (threadIdx.x==0) id[nt]=0;
    for (int t = threadIdx.x; t < nt; t += blockDim.x) id[t]=t;
    __syncthreads();

    bool more=true;
    while (__syncthreads_or(more)) {
      more = false;
      for (int t = threadIdx.x; t < nt; t += blockDim.x) {
         assert (id[t] != 999);
         assert (id[t] < nt);
         for (auto m = t+1; m<nt; ++m) {
          if (std::abs(x[m]-x[t])>1) continue;
          auto old = atomicMin(&id[m],id[t]);
          if(old!=id[t]) more=true;
          atomicMin(&id[t],old);
        }
      }
      if (threadIdx.x==0)  ++id[nt];
    }

    __syncthreads();
}


#include<iostream>
int main() {

    uint16_t x[1024]; 
    int id[1024];

    uint16_t * x_d;
    int * id_d;
    hipMalloc(&x_d, sizeof(x));
    hipMalloc(&id_d, sizeof(id));

    x[0]=1;x[1]=0;x[2]=2;
    x[3]=4;x[4]=4;x[5]=5;
    x[6]=9;x[7]=9;
    for (int i=8;i<115; ++i) x[i]=15;
    x[17]=42;x[34]=41;x[73]=42;

    int n = 115;

    hipMemcpy(x_d, x, sizeof(x),hipMemcpyHostToDevice);

    printf("size %d\n",sizeof(x));

    perm<<< 1, 64 >>>(x_d,id_d,n);

    hipMemcpy(id, id_d, sizeof(x),hipMemcpyDeviceToHost);

    std::cout << "ids " << id[0] << ' ' << id[n-1] << " loops " << id[n] << std::endl;

    return 0;

}

