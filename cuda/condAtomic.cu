
#include <hip/hip_runtime.h>
#include <cstdio>
#include<algorithm>
#include<cmath>

__global__ void bar(int n) {
    int laneId = threadIdx.x & 0x1f;
    __shared__ int res1;
    __shared__ int res2;

    res1= res2=0;
    __syncthreads();

    for (auto i=threadIdx.x; i<n; i+=blockDim.x) {
      if (laneId%3==1) atomicMax(&res1,i%laneId);
      if (laneId%3==2) atomicAdd(&res2,i%laneId);
    }

    __syncthreads();

    if (threadIdx.x==0) printf("res %d %d\n", res1,res2); 

}



int main() {

  bar<<<64,128,0>>>(154);
  hipDeviceSynchronize();
}
