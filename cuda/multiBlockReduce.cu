
#include <hip/hip_runtime.h>
#include <type_traits>

__device__ float calculatePartialSum(const float* array, unsigned int N);

template<typename T>
__device__  auto calculateTotalSum(T const * result) -> typename std::remove_volatile<T>::type
{
   __shared__ typename std::remove_volatile<T>::type ret;
   ret=0;
   __syncthreads();
   atomicAdd(&ret,result[threadIdx.x]);
   __syncthreads();
   return ret;
}

__device__ unsigned int count = 0;
__shared__ bool isLastBlockDone;
__global__ void sum(const float* array, unsigned int N,
                    float* result)
{
    // Each block sums a subset of the input array.
    float partialSum = calculatePartialSum(array, N);

    // volatile float* vresult = result;

    if (threadIdx.x == 0) {

        // Thread 0 of each block stores the partial sum
        // to global memory. The compiler will use 
        // a store operation that bypasses the L1 cache
        // since the "result" variable is declared as
        // volatile. This ensures that the threads of
        // the last block will read the correct partial
        // sums computed by all other blocks.
        result[blockIdx.x] = partialSum;

        // Thread 0 makes sure that the incrementation
        // of the "count" variable is only performed after
        // the partial sum has been written to global memory.
        __threadfence();

        // Thread 0 signals that it is done.
        unsigned int value = atomicInc(&count, gridDim.x);

        // Thread 0 determines if its block is the last
        // block to be done.
        isLastBlockDone = (value == (gridDim.x - 1));
    }

    // Synchronize to make sure that each thread reads
    // the correct value of isLastBlockDone.
    __syncthreads();

    if (isLastBlockDone) {

        // The last block sums the partial sums
        // stored in result[0 .. gridDim.x-1]
        float totalSum = calculateTotalSum(result);

        if (threadIdx.x == 0) {

            // Thread 0 of last block stores the total sum
            // to global memory and resets the count
            // varialble, so that the next kernel call
            // works properly.
            result[0] = totalSum;
            count = 0;
        }
    }
}

