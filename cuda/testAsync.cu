#include<hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include<iostream>


inline
bool cudaCheck_(const char* file, int line, const char* cmd, hipError_t result)
{
    //std::cerr << file << ", line " << line << ": " << cmd << std::endl;
    if (result == hipSuccess)
        return true;

    const char* error = hipGetErrorName(result);
    const char* message = hipGetErrorString(result);
    std::cerr << file << ", line " << line << ": " << error << ": " << message << std::endl;
    abort();
    return false;
}
#define cudaCheck(ARG) (cudaCheck_(__FILE__, __LINE__, #ARG, (ARG)))



typedef union { unsigned int n; float x; } union_t;

constexpr int maxNumOfThreads = 24;

const int bunchSize = 1024;

#include<cmath>
__global__ void kernel_foo(unsigned int n, float * py) {
   int first = blockIdx.x * blockDim.x + threadIdx.x;
   for (int i=first; i<bunchSize; i+=gridDim.x*blockDim.x) {
     union_t u; u.n = n+i; float x = u.x;
     py[i] = std::sin(x);
   }
}


void cpu_foo(unsigned int n, float * py) {
   int first = 0;
   for (int i=first; i<bunchSize; i++) {
     union_t u; u.n = n+i; float x = u.x;
     py[i] = std::sin(x);
   }
}

template <typename F>
void CUDART_CB myCallback(void * fun){
    (*(F*)(fun))();
}


void CUDART_CB aCallback(void *data){
    printf("Inside callback %d\n", *(int*)data);
}

void compare(float * yd, float * yh, float & dm) {
   int first = 0;
   for (int i=first; i<bunchSize; i++) {
     auto d = std::abs(yd[i]-yh[i]);
     dm = std::max(dm,d);
   }
}

#include<atomic>

hipStream_t streams[maxNumOfThreads];

std::atomic<int> nt(0);

void go() {
  int me = nt++;
  auto & stream = streams[me];

  float * ypD;
  float * ypH;
  float * ypC;
  float dm=0;

  cudaCheck(hipMalloc((void **)&ypD, bunchSize*sizeof(float)));
  cudaCheck(hipHostMalloc((void **)&ypH, bunchSize*sizeof(float), hipHostMallocDefault));
  ypC =(float*)::malloc(bunchSize*sizeof(float));

  union_t u; u.x=0.1f;

  kernel_foo<<<1024/128,128,0,stream>>>(u.n+me*bunchSize, ypD);

  auto k1 = [&]() {
    cpu_foo(me*bunchSize, ypC);
  };

  auto k2 = [&]() {
    compare(ypH,ypC,dm);
  };

  cudaCheck(hipMemcpyAsync(ypH, ypD, bunchSize*sizeof(float), hipMemcpyDeviceToHost, stream));
  hipLaunchHostFunc (stream, aCallback, &me);
  hipLaunchHostFunc (stream, myCallback<decltype(k1)>, &k1);
  hipLaunchHostFunc (stream, myCallback<decltype(k2)>, &k2);


  hipStreamSynchronize(stream);

  printf("max diff in %d %a\n",me,dm);

}

#include<thread>
#include<mutex>
#include<vector>

typedef std::thread Thread;
typedef std::vector<std::thread> ThreadGroup;
typedef std::mutex Mutex;
typedef std::lock_guard<std::mutex> Lock;


int main (int argc, char *argv[]) {

  printf ("Using CUDA %d\n",CUDART_VERSION);
  int cuda_device = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, cuda_device);
  printf("CUDA Capable: SM %d.%d hardware\n", deviceProp.major, deviceProp.minor);


  int nstreams = maxNumOfThreads;

  hipStream_t streams[maxNumOfThreads];


  for (int i = 0; i < nstreams; i++) {
        cudaCheck(hipStreamCreate(&(streams[i])));
  }


  ThreadGroup threads;
  threads.reserve(nstreams);

   for (int i=0; i<nstreams; ++i) {
      threads.emplace_back(go);
    }

    for (auto & t : threads) t.join();

    threads.clear();


  return 0;

}
