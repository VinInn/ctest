#include "hip/hip_runtime.h"
#include "h_ptr.h"
#include<iostream>
#include<cassert>

struct T{int a;};

struct V {
  h_ptr<T> tref;
};

__global__ void set(T * t, V* v) {
 t->a=5;
 v->tref = t;
 
}


int main() {

  relocationTable = new std::unordered_map<unsigned long long,unsigned long long>;


 T * gt;
 V * gv;
 hipMalloc(&gt,sizeof(T));
 hipMalloc(&gv,sizeof(V));
 hipMemset(gv,0,sizeof(V));
 set<<<1,1>>>(gt,gv);
 T ht;
 V hv;
 hipMemcpy(&hv,gv,sizeof(V),hipMemcpyDeviceToHost);
 assert(nullptr == hv.tref.get());
 hipMemcpy(&ht,gt,sizeof(T),hipMemcpyDeviceToHost);
 assert(nullptr == hv.tref.get());
 (*relocationTable)[(unsigned long long)gt] = (unsigned long long)(&ht);
 assert((&ht) == hv.tref.get());
 assert(5 == hv.tref.get()->a);

 hipDeviceSynchronize(); 

 // this fake cpu wf
 T ct;
 V cv;
 memset(&cv,0,sizeof(V)); 
 ct.a=5;
 cv.tref = &ct; 
 // better to be ok
 assert((&ct) == cv.tref.get());
 assert(5 == cv.tref.get()->a);


 return 0;

}
