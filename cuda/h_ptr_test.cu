#include "hip/hip_runtime.h"
#include "h_ptr.h"
#include<iostream>
#include<cassert>

struct T{int a;};

struct V {
  h_ptr<T> tref;
};

__global__ void set(T * t, V* v) {
 t->a=5;
 v->tref = t;
 
}

__global__ void get(T * t, V* v) {
 assert(v->tref.get() == t);
  assert(v->tref.get()->a == 5);
}


int main() {

  relocationTable = new std::unordered_map<unsigned long long,unsigned long long>;


 T * gt;
 V * gv, *gv2;
 hipMalloc(&gt,sizeof(T));
 hipMalloc(&gv,sizeof(V));
 set<<<1,1>>>(gt,gv);
 get<<<1,1>>>(gt,gv);

 V v2; v2.tref.setGPUptr(gt);
 hipMalloc(&gv2,sizeof(V));
 hipMemcpy(gv2,&v2,sizeof(V),hipMemcpyHostToDevice);
 get<<<1,1>>>(gt,gv2);

 T ht;
 V hv;
 hipMemcpy(&hv,gv,sizeof(V),hipMemcpyDeviceToHost);
 assert(nullptr == hv.tref.get());
 hipMemcpy(&ht,gt,sizeof(T),hipMemcpyDeviceToHost);
 assert(nullptr == hv.tref.get());
 (*relocationTable)[(unsigned long long)gt] = (unsigned long long)(&ht);
 assert((&ht) == hv.tref.get());
 assert(5 == hv.tref.get()->a);

 hipDeviceSynchronize(); 

 // this fake cpu wf
 T ct;
 V cv;
 ct.a=5;
 cv.tref = &ct; 
 // better to be ok
 assert((&ct) == cv.tref.get());
 assert(5 == cv.tref.get()->a);


 return 0;

}
