#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"


#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <chrono>

using namespace std;
using namespace std::chrono;

#define WSIZE 32
#define LOOPS 100
#define UPPER_BIT 10
#define LOWER_BIT 0

__device__ unsigned int ddata[WSIZE];
__device__ int ddata_s[WSIZE];

template <typename T, unsigned S>
inline unsigned arraysize(const T(&v)[S])
{
	return S;
}

template<typename T>
void printArray(T &arr)
{
	for (int i = 0; i < arraysize(arr); ++i)
	{
		cout << "Array[" << i << "]: " << *(arr + i) << endl;
	}
}

template<typename T>
void fillArray(T &arr)
{
	srand(time(NULL));
	for (int i = 0; i < arraysize(arr); ++i)
	{
		arr[i] = rand() % 1024;
	}
}

void print(int arr[], int n)
{
	for (int i = 0; i < n; i++)
	{
		cout << arr[i] << " ";
	}
	cout << endl;
}

template<typename T>
T findMax(T &arr)
{
	T max = 0;
	for (int i = 0; i < arraysize(arr); ++i)
	{
		if (arr[i] > max)
		{
			max = arr[i];
		}
	}
	return max;
}

__device__ int getMax(int arr[], int n)
{
	int mx = arr[0];
	for (int i = 1; i < n; i++)
		if (arr[i] > mx)
			mx = arr[i];
	return mx;
}

__device__ void countSort(int arr[], int n, int exp)
{
	int output[1024]; // Output array
	int i, count[10] = { 0 };

	// Store count of occurrences in count[]
	for (i = 0; i < n; i++)
		count[(arr[i] / exp) % 10]++;

	// Change count[i] so that count[i] now contains actual
	// position of this digit in output[]
	for (i = 1; i < 10; i++)
	{
		count[i] += count[i - 1];
	}

	// Build the output array
	for (i = n - 1; i >= 0; i--)
	{
		output[count[(arr[i] / exp) % 10] - 1] = arr[i];
		count[(arr[i] / exp) % 10]--;
	}

	// Copy the output array to arr[], so that arr[] now
	// contains sorted numbers according to current digit
	for (i = 0; i < n; i++)
		arr[i] = output[i];
}

__device__ void radixsort(int arr[], int n)
{
	// Find the maximum number to know number of digits
	int m = getMax(arr, n);

	// Do counting sort for every digit. Note that instead
	// of passing digit number, exp is passed. exp is 10^i
	// where i is current digit number
	for (int exp = 1; m / exp > 0; exp *= 10)
		countSort(arr, n, exp);
}

__global__ void serialRadix()
{
	radixsort(ddata_s, WSIZE);
	__syncthreads();
}

__global__ void parallelRadix()
{
	// This data in shared memory
	__shared__ volatile unsigned int sdata[WSIZE * 2];

	// Load from global into shared variable
	sdata[threadIdx.x] = ddata[threadIdx.x];

	unsigned int bitmask = 1 << LOWER_BIT;
	unsigned int offset = 0;
	// -1, -2, -4, -8, -16, -32, -64, -128, -256,...
	unsigned int thrmask = 0xFFFFFFFFU << threadIdx.x;
	unsigned int mypos;

	// For each LSB to MSB
	for (int i = LOWER_BIT; i <= UPPER_BIT; i++)
	{
		unsigned int mydata = sdata[((WSIZE - 1) - threadIdx.x) + offset];
		unsigned int mybit = mydata&bitmask;
		// Get population of ones and zeroes
		unsigned int ones = __ballot(mybit);
		unsigned int zeroes = ~ones;
		// Switch ping-pong buffers
		offset ^= WSIZE;

		// Do zeroes, then ones
		if (!mybit)
		{
			mypos = __popc(zeroes&thrmask);
		}
		else  {      // Threads with a one bit
			// Get my position in ping-pong buffer
			mypos = __popc(zeroes) + __popc(ones&thrmask);
		}

		// Move to buffer  (or use shfl for cc 3.0)
		sdata[mypos - 1 + offset] = mydata;
		// Repeat for next bit
		bitmask <<= 1;
	}
	// Put results to global
	ddata[threadIdx.x] = sdata[threadIdx.x + offset];
}

int main() {

	/* Parallel Radix Sort */

	unsigned int hdata[WSIZE];
	float totalTime = 0;

	for (int lcount = 0; lcount < LOOPS; lcount++)
	{
		srand(time(NULL));
		// Array elements have value in range of 1024
		unsigned int range = 1U << UPPER_BIT;

		// Fill array with random elements
		// Range = 1024
		for (int i = 0; i < WSIZE; i++)
		{
			hdata[i] = i;
		}

		// Copy data from host to device
		hipMemcpyToSymbol(HIP_SYMBOL(ddata), hdata, WSIZE * sizeof(unsigned int));

		// Execution time measurement, that point starts the clock
		high_resolution_clock::time_point t1 = high_resolution_clock::now();
		parallelRadix <<< 1, WSIZE >>>();
		// Make kernel function synchronous
		hipDeviceSynchronize();
		// Execution time measurement, that point stops the clock
		high_resolution_clock::time_point t2 = high_resolution_clock::now();

		// Execution time measurement, that is the result
		auto duration = duration_cast<milliseconds>(t2 - t1).count();

		// Summination of each loops' execution time
		totalTime += (float)duration / 1000.00;

		// Copy data from device to host
		hipMemcpyFromSymbol(hdata, HIP_SYMBOL(ddata), WSIZE * sizeof(unsigned int));
	}

	printf("Parallel Radix Sort:\n");
	printf("Array size = %d\n", WSIZE * LOOPS);
	printf("Time elapsed = %fseconds\n", totalTime);

	/* Serial Radix Sort */

	unsigned int hdata_s[WSIZE];
	totalTime = 0;

	for (int lcount = 0; lcount < LOOPS; lcount++)
	{
		srand(time(NULL));
		// Array elements have value in range of 1024
		unsigned int range = 1U << UPPER_BIT;

		// Fill array with random elements
		// Range = 1024
		for (int i = 0; i < WSIZE; i++)
		{
			hdata_s[i] = i;
		}

		// Copy data from host to device
		hipMemcpyToSymbol(HIP_SYMBOL(ddata_s), hdata_s, WSIZE * sizeof(unsigned int));

		// Execution time measurement, that point starts the clock
		high_resolution_clock::time_point t1 = high_resolution_clock::now();
		serialRadix <<< 1, 1 >>>();
		// Make kernel function synchronous
		hipDeviceSynchronize();
		// Execution time measurement, that point stops the clock
		high_resolution_clock::time_point t2 = high_resolution_clock::now();

		// Execution time measurement, that is the result
		auto duration = duration_cast<milliseconds>(t2 - t1).count();

		// Summination of each loops' execution time
		totalTime += (float)duration / 1000.00;

		// Copy data from device to host
		hipMemcpyFromSymbol(hdata_s, HIP_SYMBOL(ddata_s), WSIZE * sizeof(unsigned int));
	}

	printf("\nSerial Radix Sort:\n");
	printf("Array size = %d\n", WSIZE * LOOPS);
	printf("Time elapsed = %fseconds\n\n", totalTime);

	return 0;
}

