#include "cudaMemoryPool.h"

#include "FastPoolAllocator.h"

FastPoolAllocatorImpl<CudaDeviceAlloc>  devicePool(1024);
FastPoolAllocatorImpl<CudaHostAlloc>  hostPool(1024);


namespace memoryPool {
  namespace cuda {

    void dumpStat() {
       std::cout << "device pool" << std::endl;
       devicePool.dumpStat();
       std::cout << "host pool" << std::endl;
       hostPool.dumpStat();

    }


   FastPoolAllocator * getPool(Where where) {
      return onDevice==where ?  (FastPoolAllocator *)(&devicePool) : (FastPoolAllocator *)(&hostPool);
   }

    struct Payload {
      FastPoolAllocator * pool;
      std::vector<int> buckets;
    };

    // generic callback
    void CUDART_CB freeCallback(void * p){
      auto payload = (Payload*)(p);
      auto & pool = *(payload->pool);
      auto const & buckets = payload->buckets;
        std::cout << "do free " << buckets.size();
        if (!buckets.empty()) std::cout  << ' ' << buckets.front() << ' ' << buckets.back();
        std::cout << std::endl;
        for (auto i :  buckets) {
          pool.free(i);
        }
      delete payload;
    }

    // allocate either on current device or on host
    std::pair<void *,int> alloc(uint64_t size, FastPoolAllocator & pool) {
       int i = pool.alloc(size);
       void * p = pool.pointer(i);
       return std::pair<void *,int>(p,i);
    }

    // schedule free
    void free(hipStream_t stream, std::vector<int> buckets, FastPoolAllocator & pool) {
      // free
      std::cout << "schedule free " << buckets.size() << ' ';
      if (!buckets.empty()) std::cout << buckets[0]; 
      std::cout << std::endl;
      auto payload = new Payload{&pool, std::move(buckets)};
      hipLaunchHostFunc (stream, freeCallback, payload);
    }

  }
}
