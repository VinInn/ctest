#include "cudaMemoryPool.h"

#include "SimplePoolAllocator.h"


#include<hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

struct CudaDeviceAlloc {

  using Pointer = void *;

  static Pointer alloc(size_t size) { Pointer p=nullptr; auto err = hipMalloc(&p,size); return err==hipSuccess ? p : nullptr;}
  static void free(Pointer ptr) { hipFree(ptr); }

};

struct CudaHostAlloc {

  using Pointer = void *;

  static Pointer alloc(size_t size) { Pointer p=nullptr; auto err = hipHostMalloc(&p,size); return err==hipSuccess ? p : nullptr;}
  static void free(Pointer ptr) { hipHostFree(ptr); }

};



SimplePoolAllocatorImpl<CudaDeviceAlloc>  devicePool(1024);
SimplePoolAllocatorImpl<CudaHostAlloc>  hostPool(1024);


namespace memoryPool {
  namespace cuda {

    void dumpStat() {
       std::cout << "device pool" << std::endl;
       devicePool.dumpStat();
       std::cout << "host pool" << std::endl;
       hostPool.dumpStat();

    }


   SimplePoolAllocator * getPool(Where where) {
      return onDevice==where ?  (SimplePoolAllocator *)(&devicePool) : (SimplePoolAllocator *)(&hostPool);
   }

    struct Payload {
      SimplePoolAllocator * pool;
      std::vector<int> buckets;
    };

    // generic callback
    void CUDART_CB freeCallback(void * p){
      auto payload = (Payload*)(p);
      auto & pool = *(payload->pool);
      auto const & buckets = payload->buckets;
        std::cout << "do free " << buckets.size();
        if (!buckets.empty()) std::cout  << ' ' << buckets.front() << ' ' << buckets.back();
        std::cout << std::endl;
        for (auto i :  buckets) {
          pool.free(i);
        }
      delete payload;
    }

    // allocate either on current device or on host
    std::pair<void *,int> alloc(uint64_t size, SimplePoolAllocator & pool) {
       int i = pool.alloc(size);
       void * p = pool.pointer(i);
       return std::pair<void *,int>(p,i);
    }

    // schedule free
    void free(hipStream_t stream, std::vector<int> buckets, SimplePoolAllocator & pool) {
      // free
      std::cout << "schedule free " << buckets.size() << ' ';
      if (!buckets.empty()) std::cout << buckets[0]; 
      std::cout << std::endl;
      auto payload = new Payload{&pool, std::move(buckets)};
      hipLaunchHostFunc (stream, freeCallback, payload);
    }

  }
}
