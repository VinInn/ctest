#include "cudaMemoryPool.h"

#include<atomic>
#include<thread>
#include<mutex>
#include<chrono>

int main() {


  auto start = std::chrono::high_resolution_clock::now();

  const int NUMTHREADS=24;

  printf ("Using CUDA %d\n",CUDART_VERSION);
  int cuda_device = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, cuda_device);
  printf("CUDA Capable: SM %d.%d hardware\n", deviceProp.major, deviceProp.minor);


  hipStream_t streams[NUMTHREADS];


  for (int i = 0; i < NUMTHREADS; i++) {
     hipStreamCreate(&(streams[i]));
  }

  memoryPool::cuda::dumpStat();

  auto & stream = streams[0]; 

  {
    auto pd = memoryPool::cuda::make_unique<int>(20,stream,memoryPool::onDevice);
    auto ph = memoryPool::cuda::make_unique<int>(20,stream,memoryPool::onHost);

    memoryPool::cuda::dumpStat();
  }


  {
     memoryPool::Deleter devDeleter(std::make_shared<memoryPool::cuda::BundleDelete>(stream,memoryPool::onDevice));
     memoryPool::Deleter hosDeleter(std::make_shared<memoryPool::cuda::BundleDelete>(stream,memoryPool::onHost));

     auto p0 = memoryPool::cuda::make_unique<int>(20,devDeleter);
     auto p1 = memoryPool::cuda::make_unique<double>(20,devDeleter);
     auto p2 = memoryPool::cuda::make_unique<bool>(20,devDeleter);
     auto p3 = memoryPool::cuda::make_unique<int>(20,devDeleter);

     auto hp0 = memoryPool::cuda::make_unique<int>(20,hosDeleter);
     auto hp1 = memoryPool::cuda::make_unique<double>(20,hosDeleter);
     auto hp2 = memoryPool::cuda::make_unique<bool>(20,hosDeleter);
     auto hp3 = memoryPool::cuda::make_unique<int>(20,hosDeleter);

     memoryPool::cuda::dumpStat();
  }

  hipStreamSynchronize(stream);
  memoryPool::cuda::dumpStat();

  return 0;
}
