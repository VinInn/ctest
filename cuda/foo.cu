#include "hip/hip_runtime.h"
#include <cstdio>
__global__ 
void foo() {
   printf("Hello\n");
}

#include<hip/hip_runtime.h>
#include "cudaCheck.h"

void fooWrapper() {

  foo<<<1,1>>>();
  cudaCheck(hipGetLastError());
  hipDeviceSynchronize();
 

}


void docheck() {
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
}
