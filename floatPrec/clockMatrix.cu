#include "hip/hip_runtime.h"
// /usr/local/cuda/bin/nvcc -gencode arch=compute_75,code=sm_75 -O3 --expt-relaxed-constexpr clock.cu -DCLOCK -DFLOAT=float
#include<cstdint>
#include<cmath>
#include<random>
#include<cstdio>

#include "Matrix.h"
#include "TwoFloat.h"

// generate matrices
template <typename M, typename Eng>
void genMatrix(M& m, Eng & eng) {
  // using T = typename std::remove_reference<decltype(m(0, 0))>::type;
  int n = M::kRows;
  std::uniform_real_distribution<float> rgen(0., 1.);

  // generate first diagonal elemets
  for (int i = 0; i < n; ++i) {
    float maxVal = i * 1.e10 / (n - 1) + 1;  // max condition is 10^10
    m(i, i) = maxVal * rgen(eng) + 1.e-9;
  }
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < i; ++j) {
      float v = 0.3f * std::sqrt(float(m(i, i) * m(j, j)));  // this makes the matrix pos defined
      m(i, j) = v * rgen(eng) + 1.e-9;;
      // m(j, i) = m(i, j);
    }
  }
}



using Float = FLOAT;

#if defined(TWOF)
using MM5 = MatrixSym<TwoFloat<Float>,5>;
#else
using MM5 = MatrixSym<Float,5>;
#endif

// Type your code here, or load an example.
__global__ void square(MM5 * array,  int64_t * tt, int64_t * tg, int n) {
     int maxIter = 100000;
     __shared__ uint64_t gstart, gend;
     uint64_t start, end;
     int tid = blockDim.x * blockIdx.x + threadIdx.x;

     auto m1 = array[tid];
     MM5 m2;

     if (tid==0) {
#ifdef CLOCK
      gstart = clock64();
#else
      // Record start time
      asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(gstart));
#endif
     }
     __syncthreads();
#ifdef CLOCK     
    auto s = clock64();
#else    
    // Record start time
    asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(start));
#endif
     if (tid<n) {
       for (int kk=0; kk<maxIter; ++kk) {
          invert55(m1,m2);
          invert55(m2,m1);
       }
    // Record end time 
#ifdef CLOCK
       tt[tid] = clock64() -s;
#else
   asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(end));
   tt[tid] = end - start;
#endif
    }

    __syncthreads();
    if (tid==0) {
 #ifdef CLOCK
      *tg = clock64() -gstart;
#else
     asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(gend));
     *tg = gend - gstart;
#endif
   }
   array[tid] = m1;
}

#include<iostream>

int main(int argc, char** argv) {

  constexpr int nB = 1;
  constexpr int nT = 128;

  constexpr int n = nB*nT;
  MM5 * a;
  int64_t * tt;
  int64_t * tg;

   MM5 m0[n];

  hipMallocManaged(&a, n*sizeof(MM5));
  hipMallocManaged(&tt, n*sizeof(int64_t));
  hipMallocManaged(&tg, sizeof(int64_t));

  std::mt19937 eng;
  for (int i=0; i<n; ++i) genMatrix(a[i], eng);
  for (int i=0; i<n; ++i) m0[i] = a[i];

  for (int i=0; i<n; ++i) tt[i]=0;
  *tg=0;
  square<<<nB,nT,0,0>>>(a,tt,tg,n);
  hipDeviceSynchronize();

  Float maxOn=0;
  Float maxOff=0;
  int ns = 5;
  for (int i=0; i<n; ++i) {
    auto const & m1 = m0[i];
    auto const & m3 = a[i];
    for (int i=0; i<ns; ++i)
#if defined(TWOF)
     maxOn = std::max(maxOn,std::abs( ((m3(i,i)-m1(i,i))/m1(i,i)).hi() ));
#else
      maxOn = std::max(maxOn,std::abs( (m3(i,i)-m1(i,i))/m1(i,i) ));
#endif
    for (int i = 0; i < ns; ++i) {
      for (int j = 0; j < i; ++j) {
#if defined(TWOF)
         maxOff = std::max(maxOn,std::abs( ((m3(i,j)-m1(i,i))/m1(i,j)).hi() ));
#else
         maxOff = std::max(maxOn,std::abs( (m3(i,j)-m1(i,i))/m1(i,j) ));
#endif
      }
    }
  }
  std::cout << maxOn << ' ' << maxOff << std::endl;

  for (int i=0; i<n; ++i) std::cout << tt[i] <<  ' ';
  std::cout << '\n' << *tg << std::endl;

  hipFree(a);
  hipFree(tt);
  hipFree(tg);

  return 0;
}
